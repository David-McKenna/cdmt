#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <time.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <getopt.h>
#include <limits.h>
#include <omp.h>

#include "udpPacketManager/lofar_udp_reader.h"
#include "udpPacketManager/lofar_udp_misc.h"

#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10


// Struct for header information
struct header {
  int nchan,nbit=0,nsub,tel=11;
  double tstart,tsamp,fch1,foff,fcen,bwchan;
  double src_raj,src_dej;
  char source_name[80];
  char rawfname[4][1024];
};



struct header read_sigproc_header(char *fname, char *dataname, int ports);
void get_channel_chirp(double fcen,double bw,float dm,int nchan,int nbin,int nsub,hipfftComplex *c);
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s);
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b);
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale);
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2);
__global__ void unpack_and_padd_first_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2);
__global__ void padd_next_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2);
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny);
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c);
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2);
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd);
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
__global__ void decimate(float *z,int ndec,int nchan,int nblock,int nsum,float *cz);
void write_to_disk_float(float* outputArray, FILE* outputFile, int nsamples, hipEvent_t waitEvent);
void write_to_disk_char(unsigned char* outputArray, FILE* outputFile, int nsamples, hipEvent_t waitEvent);
void write_filterbank_header(struct header h,FILE *file);
int reshapeRawUdp(lofar_udp_reader *reader);
long  __inline__ beamformed_packno(unsigned int timestamp, unsigned int sequence);

extern "C"  {
  int lofar_udp_reader_step(lofar_udp_reader *reader);
  lofar_udp_reader* lofar_udp_meta_file_reader_setup(FILE **inputFiles, const int numPorts, const int replayDroppedPackets, const int processingMode, const int verbose, const long packetsPerIteration, const long startingPacket, const long packetsReadMax, const int compressedReader);
  int lofar_udp_file_reader_reuse(lofar_udp_reader *reader, const long startingPacket, const long packetsReadMax);
}
// Usage
void usage()
{
  printf("cdmt -v -c -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -f <number of FFTs per operation> -o <outputname> -s <sigproc header location> -p <port nums> <fil prefix>\n\n");
  printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in raw udp format.\n");
  printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
  printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
  printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
  printf("-o <outputname>           Output filename [default: cdmt]\n");
  printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
  printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");
  printf("-s <packets>       Number of packets to skip in the filterbank before stating processing [integer, default: 0]\n");
  printf("-r <packets>       Number of packets to read in total from the -s offset [integer, default: length of file]\n");
  printf("-m <sigproc header location>  Sigproc header to read metadata from [default: fil prefix.sigprochdr]\n");
  printf("-f <FFTs per op> Number of FFTs to execute per cuFFT call [default: 128]\n");
  printf("-a               Disable redigitisation; output float32 [default: false]\n");
  printf("-c <num chan>    Channelisation Factor [default: 8]\n");

  return;
}

int main(int argc,char *argv[])
{
  int i,j,nsamp,nfft,mbin,nvalid,nchan=8,nbin=65536,noverlap=2048,nsub=20,ndm,ndec=1;
  int idm,iblock,nread_tmp,nread,mchan,msamp,mblock,msum=1024;
  char *header,*udpbuf[4],*dudpbuf[4];
  FILE *file;
  unsigned char **cbuf[2],*dcbuf;
  float **cbuff[2], *dcbuff;
  float *fbuf,*dfbuf;
  float *bs1,*bs2,*zavg,*zstd;
  hipfftComplex *cp1,*cp2,*dc,*cp1p,*cp2p;
  hipfftHandle ftc2cf,ftc2cb;
  int idist,odist,iembed,oembed,istride,ostride;
  dim3 blocksize,gridsize;
  struct header hdr;
  clock_t startclock;
  float *dm,*ddm,dm_start,dm_step;
  char fname[128],fheader[1024],*udpfname,sphdrfname[1024] = "",obsid[128]="cdmt";
  int bytes_read;
  long int ts_read=LONG_MAX,ts_skip=0;
  long int total_ts_read=0;
  int part=0,device=0,verbose=0,nforward=128,redig=1,ports=4;
  int arg=0;
  FILE **outfile;

  lofar_udp_reader *reader;

  // Read options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"ac:p:f:d:D:ho:b:N:n:s:r:m:t:"))!=-1) {
      switch (arg) {
  
      case 'n':
  noverlap=atoi(optarg);
  break;

      case 'N':
  nbin=atoi(optarg);
  break;

      case 'b':
  ndec=atoi(optarg);
  break;

      case 'o':
  strcpy(obsid,optarg);
  break;

      case 'D':
  device=atoi(optarg);
  break;

      case 's':
  ts_skip=atol(optarg) / 16;
  break;
  
      case 'r':
  ts_read=atol(optarg);
  break;

      case 'd':
  sscanf(optarg,"%f,%f,%d",&dm_start,&dm_step,&ndm);
  break;

      case 'm':
  strcpy(sphdrfname,optarg);
  break;

      case 'f':
  nforward=atoi(optarg);
  break;
  
      case 'a':
  redig=0;
  break;

      case 'c':
  nchan=atoi(optarg);
  break;

      case 'h':
  usage();
  return 0;

  return 0;
      }
    }
  } else {
    printf("Unknown option '%c'\n", arg);
    usage();
    return 0;
  }
  udpfname=argv[optind];

  // Sanity checks to avoid voids in output filterbank
  if (nbin % 8 != 0) {
    fprintf(stderr, "ERROR: nbin must be disible by 8 (currently %d, remainder %d). Exiting.\n", nbin, nbin % 8);
    exit(1);
  }
  if ( (nforward * (nbin-2*noverlap)) % 8 != 0 ) {
    fprintf(stderr, "ERROR: Valid data length must be divisible by 8 (currently %d, remainer %d). Exiting.", nbin-2*noverlap, (nbin-2*noverlap) % 8);
    exit(1);
  }

  if ((nforward * (nbin-2*noverlap) / 8) % 1024 != 0) {
    fprintf(stderr, "ERROR: Interal sum cannot proceed; valid samples must be divisible by 1024 (currently %d, remainder %d).\n", (nforward * (nbin-2*noverlap) / 8), (nforward * (nbin-2*noverlap) / 8) % 1024);
    fprintf(stderr, "Consider using %d or %d as your forward FFT size next time. Exiting.\n", 64 * ((nforward * (nbin-2*noverlap) / 8) - (nforward * (nbin-2*noverlap) / 8) % 1024) / 1024 + 2 * noverlap,
                                                                                   64 * ((nforward * (nbin-2*noverlap) / 8) + (1024  - (nforward * (nbin-2*noverlap) / 8) % 1024)) / 1024 + 2 * noverlap);
    exit(1);
  }
  

  if (strcmp(sphdrfname, "") == 0) {
    sprintf(sphdrfname, "%s.sigprochdr", udpfname);
  }

  FILE* inputFiles[4];
  int compressedInput = 0;
  char tmpfname[1024] = "";
  if (strstr(udpfname, "zst") != NULL) compressedInput = 1;


 
  // Read sigproc header
  hdr = read_sigproc_header(sphdrfname, udpfname, ports);
  const double stg1 = (1.0 / 2.41e-4) *  abs(pow((double) hdr.fch1 + hdr.nsub * hdr.foff + hdr.foff *0.5,-2.0) - pow((double) hdr.fch1 + hdr.nsub * hdr.foff - hdr.foff *0.5, -2.0));
  const int overlapCheck = (int) (stg1 * (dm_start + dm_step * ndm)/ hdr.tsamp);
  if (overlapCheck > nbin) {
    fprintf(stderr, "WARNING: The size of your FFT bin is too short for the given DMs and frequencies. Given bin size: %d, Suggested minimum bin size: %d\n", nbin, overlapCheck);
  } else if (overlapCheck > noverlap) {
    fprintf(stderr, "WARNING: The size of your FFT overlap is too short for the given maximum DM. Given overlap: %d, Suggested minimum overlap: %d.\n", noverlap, overlapCheck);
  }


  for (int i = 0; i < 4; i++) {
    sprintf(tmpfname, udpfname, i);
    printf("Opening %s...\n", tmpfname);

    inputFiles[i] = fopen(tmpfname, "r");

    if (inputFiles[i] == NULL) {
      printf("Input file failed to open (null pointer)\n");
    }

    strcpy(hdr.rawfname[i],tmpfname);
  }
  // Read the number of subbands
  nsub=hdr.nsub;
  double timeOffset = hdr.tsamp;

  // Adjust header for filterbank format
  hdr.tsamp*=nchan*ndec;
  hdr.nchan=nsub*nchan;
  if (redig) hdr.nbit=8;
  else hdr.nbit=32;
  hdr.fch1=hdr.fcen+0.5*hdr.nsub*hdr.bwchan-0.5*hdr.bwchan/nchan;
  hdr.foff=-fabs(hdr.bwchan/nchan);


  // Data size
  nvalid=nbin-2*noverlap;
  nsamp=nforward*nvalid;
  nfft=(int) ceil(nsamp/(float) nvalid);
  mbin=nbin/nchan; // nbin must be evenly divisible by 8
  mchan=nsub*nchan;
  msamp=nsamp/nchan; // nforward * nvalid must be divisble by 8
  mblock=msamp/msum; // nforward * nvalid / 8 must be disible by 1024

  long unsigned int bytesUsed = sizeof(hipfftComplex) * nbin * nfft * nsub * 4 + sizeof(hipfftComplex) * nbin *nsub * ndm + sizeof(float) * mblock * mchan * 2 + sizeof(char) * nsamp * nsub * 4 + sizeof(float) * nsamp * nsub + redig * msamp * mchan / ndec - (redig - 1) * 4 * msamp * mchan * ndec;
  printf("We anticipate %ld GB (%ld bytes) to be allocated on the GPU.\n", bytesUsed >> 30, bytesUsed);


  const long int packetGulp = nsamp / 16;
  reader = lofar_udp_meta_file_reader_setup(inputFiles, ports, 1, 11, 1, packetGulp, (long) -1, LONG_MAX, compressedInput);
  if (reader == NULL) {
    fprintf(stderr, "Failed to generate LOFAR UDP Reader, exiting.\n");
    exit(1);
  }

  if (ts_skip > 0) {
      if (int returnVal = lofar_udp_file_reader_reuse(reader, reader->meta->lastPacket + ts_skip, -1) > 0) {
        fprintf(stderr, "Error re-initialising reader for %ld packets skipped (error %d), exiting.\n", ts_skip, returnVal);
        exit(1);
      }

      printf("Skipped %ld time stemps.\n", ts_skip * 16 );
  }

  hdr.tstart = lofar_get_packet_time_mjd(reader->meta->inputData[0]);
  if (verbose) printf("lofar_udp_reader Generated successfully.\n");

  // Set device
  checkCudaErrors(hipSetDevice(device));

  // Generate streams for asyncrnous operations
  hipStream_t streams[3];
  for (i = 0; i < 3; i++)
    checkCudaErrors(hipStreamCreate(&(streams[i])));

  // Create 2 events; one which blocks execution (preventing new data reads) and the other waiting for compute to finish.
  hipEvent_t events[3];
  hipEventCreateWithFlags(&(events[0]), hipEventBlockingSync & hipEventDisableTiming);
  hipEventCreateWithFlags(&(events[1]), hipEventDisableTiming);
  hipEventCreateWithFlags(&(events[2]), hipEventDisableTiming);

  hipEvent_t dmWriteEvents[2][ndm];
  for (i =0; i < ndm; i++)
    for (j = 0; j < 2; j++)
      hipEventCreateWithFlags(&(dmWriteEvents[j][i]), hipEventBlockingSync & hipEventDisableTiming);

  // DMcK: cuFFT docs say it's best practice to plan before allocating memory
  // cuda-memcheck fails initialisation before this block is run?
  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub));
  hipfftSetStream(ftc2cf,streams[0]);
  hipDeviceSynchronize();
  // Total malloc (FFT forward)

  // Generate FFT plan (batch in-place backward FFT)
  idist=mbin;  odist=mbin;  iembed=mbin;  oembed=mbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub));
  hipfftSetStream(ftc2cb,streams[0]);
  hipDeviceSynchronize();
  // Total malloc (backward)

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,  (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2,  (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp1p, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2p, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));

  // Allocate device memory for chirp
  checkCudaErrors(hipMalloc((void **) &dc, (size_t) sizeof(hipfftComplex)*nbin*nsub*ndm));

  if (redig) {
    // Allocate device memory for block sums
    checkCudaErrors(hipMalloc((void **) &bs1, (size_t) sizeof(float)*mblock*mchan));
    checkCudaErrors(hipMalloc((void **) &bs2, (size_t) sizeof(float)*mblock*mchan));

    // Allocate device memory for channel averages and standard deviations
    checkCudaErrors(hipMalloc((void **) &zavg, (size_t) sizeof(float)*mchan));
    checkCudaErrors(hipMalloc((void **) &zstd, (size_t) sizeof(float)*mchan));
  }

  // Allocate memory for redigitized output and header
  header=(char *) malloc(sizeof(char)*HEADERSIZE);
  for (i=0;i<4;i++) {
    udpbuf[i]= reader->meta->outputData[i];
    checkCudaErrors(hipMalloc((void **) &dudpbuf[i], (size_t) sizeof(char)*nsamp*nsub));
  }

  // Allocate output buffers
  fbuf=(float *) malloc(sizeof(float)*nsamp*nsub);
  checkCudaErrors(hipMalloc((void **) &dfbuf, (size_t) sizeof(float)*nsamp*nsub));
  
  if (redig) {
    for (i = 0; i < ndm; i++)
      for (j = 0; j < 2; j++)
        cbuf[j][i]=(unsigned char *) malloc(sizeof(unsigned char)*msamp*mchan/ndec);
    checkCudaErrors(hipMalloc((void **) &dcbuf, (size_t) sizeof(unsigned char)*msamp*mchan/ndec));
  } else {
    for (i = 0; i < ndm; i++)
      for (j = 0; j < 2; j++)
        cbuff[j][i] = (float *) malloc(sizeof(float)*msamp*mchan/ndec);
    if (ndec > 1) checkCudaErrors(hipMalloc((void **) &dcbuff, (size_t) sizeof(float)*msamp*mchan/ndec));
  }


  // Allocate DMs and copy to device
  dm=(float *) malloc(sizeof(float)*ndm);
  for (idm=0;idm<ndm;idm++)
    dm[idm]=dm_start+(float) idm*dm_step;
  checkCudaErrors(hipMalloc((void **) &ddm, (size_t) sizeof(float)*ndm));
  checkCudaErrors(hipMemcpy(ddm,dm,sizeof(float)*ndm,hipMemcpyHostToDevice));

  // Allow memory alloation/copy actions to finish before processing
  hipDeviceSynchronize();
  if (verbose) printf("Malloc complete.\n");
  // Compute chirp
  blocksize.x=32; blocksize.y=32; blocksize.z=1;
  gridsize.x=nsub/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=ndm/blocksize.z+1;
  compute_chirp<<<gridsize,blocksize>>>(hdr.fcen,nsub*hdr.bwchan,ddm,nchan,nbin,nsub,ndm,dc);
  if (verbose) printf("Chirp calculated.\n");
  // Write temporary filterbank header
  file=fopen("/tmp/header.fil","w");
  if (file == NULL) {
    fprintf(stderr, "ERROR: Unable to open /tmp/header.fil to write temporary header; exiting.\n");
    exit(1);
  }
  write_filterbank_header(hdr,file);
  fclose(file);
  file=fopen("/tmp/header.fil","r");
  if (file == NULL) {
    fprintf(stderr, "ERROR: Unable to re-open /tmp/header.fil to read temporary header length; exiting.\n");
    exit(1);
  }
  bytes_read=fread(fheader,sizeof(char),1024,file);
  fclose(file);
  
  // Format file names and open
  outfile=(FILE **) malloc(sizeof(FILE *)*ndm);
  for (idm=0;idm<ndm;idm++) {
    sprintf(fname,"%s_cDM%06.2f_P%03d.fil",obsid,dm[idm],part);

    outfile[idm]=fopen(fname,"w");
    if (outfile[idm] == NULL) {
      fprintf(stderr, "Unable to open output file %s, exiting.\n", fname);
      exit(1);
    }
  }
  if (verbose) printf("Output header generated successfully.\n");
  // Write headers
  for (idm=0;idm<ndm;idm++) {
    // Send header
    fwrite(fheader,sizeof(char),bytes_read,outfile[idm]);
  }
  if (verbose) printf("Output header written successfully.\n");
  if (verbose) printf("Starting processing loop.\n\n");

  // Loop over input file contents
  double timeInSeconds = 0.0;
  nread = INT_MAX;

  omp_lock_t readingLock;
  omp_init_lock(&readingLock);

  // Skip the first noverlap samples as they are 0'd
  int writeOffset = noverlap * 2;
  #pragma omp parallel for ordered schedule(static, 1) private(writeOffset)
  for (iblock=0;;iblock++) {
    #pragma omp cancellation point parallel

    if (iblock != 0)
      writeOffset = 0;
    else
      writeOffset = noverlap * 2;

    // Ge tthe current stream from block iteration
    int streamIdx = iblock % 2;
    hipStream_t stream = streams[streamIdx];
    // Read block
    #pragma omp ordered single
    {

      hipEventSynchronize(events[0]);
      startclock=clock();
      nread_tmp = reshapeRawUdp(reader);

      if (nread > nread_tmp) {
        nread = nread_tmp;
      }

      if (nread==0) {
        printf("No data read from last file; assuming EOF, finishng up.\n");
        #pragma omp cancel parallel for
      } else if (iblock != 0 && nread < nread_tmp) {
        printf("Received less data than expected; we may have parsed out of order data or we are nearing the EOF.\n");
      }

      // Count up the total bytes read
      total_ts_read += nread;
      printf("Block: %d: Read %ld MB in %.2f s\n",iblock,sizeof(char)*nread*nsub*4/(1<<20),(float) (clock()-startclock)/CLOCKS_PER_SEC);

      omp_set_lock(&readingLock);
      hipStreamWaitEvent(stream, events[1], 0);
      // Copy buffers to device
      startclock=clock();
      for (i=0;i<4;i++)
        checkCudaErrors(hipMemcpyAsync(dudpbuf[i],udpbuf[i],sizeof(char)*nread*nsub,hipMemcpyHostToDevice,stream));

      checkCudaErrors(hipEventRecord(events[0], stream));
    }
    // Unpack data and padd data
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
    if (iblock > 0) {
      unpack_and_padd<<<gridsize,blocksize,0,stream>>>(dudpbuf[0],dudpbuf[1],dudpbuf[2],dudpbuf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);
    }
    else {
      unpack_and_padd_first_iteration<<<gridsize,blocksize,0,stream>>>(dudpbuf[0],dudpbuf[1],dudpbuf[2],dudpbuf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);;
    }

    // Perform FFTs
    hipfftSetStream(ftc2cf, stream);
    hipfftSetStream(ftc2cb, stream);
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1p,(hipfftComplex *) cp1p,HIPFFT_FORWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2p,(hipfftComplex *) cp2p,HIPFFT_FORWARD));

    // Swap spectrum halves for large FFTs
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft*nsub/blocksize.y+1; gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize,0,stream>>>(cp1p,cp2p,nbin,nfft*nsub);

    // Loop over dms
    for (idm=0;idm<ndm;idm++) {

      // Perform complex multiplication of FFT'ed data with chirp
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=nbin*nsub/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=1;
      PointwiseComplexMultiply<<<gridsize,blocksize,0,stream>>>(cp1p,dc,cp1,nbin*nsub,nfft,idm,1.0/(float) nbin);
      PointwiseComplexMultiply<<<gridsize,blocksize,0,stream>>>(cp2p,dc,cp2,nbin*nsub,nfft,idm,1.0/(float) nbin);
      
      // end of cp1p/cp2p needed

      if (idm == ndm - 1) {
        checkCudaErrors(hipEventRecord(events[2], stream));
        hipStreamWaitEvent(streams[2], events[2], 0);

        blocksize.x=32; blocksize.y=32; blocksize.z=1;
        gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
        padd_next_iteration<<<gridsize,blocksize,0,streams[2]>>>(dudpbuf[0],dudpbuf[1],dudpbuf[2],dudpbuf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);
        checkCudaErrors(hipEventRecord(events[1], streams[2]));
        omp_unset_lock(&readingLock);
      }
      // Swap spectrum halves for small FFTs
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan*nfft*nsub/blocksize.y+1; gridsize.z=1;
      swap_spectrum_halves<<<gridsize,blocksize,0,stream>>>(cp1,cp2,mbin,nchan*nfft*nsub);
      
      // Perform FFTs
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
      
      hipStreamWaitEvent(stream, dmWriteEvents[streamIdx][idm-1 > -1 ? idm-1 : 0], 0);
      // Detect data
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=nfft/blocksize.z+1;
      transpose_unpadd_and_detect<<<gridsize,blocksize,0,stream>>>(cp1,cp2,mbin,nchan,nfft,nsub,noverlap/nchan,nread/nchan,dfbuf);
      
      // end of cp1/cp2 needed
      
      if (redig) {
        // Compute block sums for redigitization
        blocksize.x=32; blocksize.y=32; blocksize.z=1;
        gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
        compute_block_sums<<<gridsize,blocksize,0,stream>>>(dfbuf,mchan,mblock,msum,bs1,bs2);
        
        // Compute channel stats
        blocksize.x=32; blocksize.y=1; blocksize.z=1;
        gridsize.x=mchan/blocksize.x+1; gridsize.y=1; gridsize.z=1;
        compute_channel_statistics<<<gridsize,blocksize,0,stream>>>(mchan,mblock,msum,bs1,bs2,zavg,zstd);

        // Redigitize data to 8bits
        blocksize.x=32; blocksize.y=32; blocksize.z=1;
        gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
        if (ndec==1)
    redigitize<<<gridsize,blocksize,0,stream>>>(dfbuf,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);
        else
    decimate_and_redigitize<<<gridsize,blocksize,0,stream>>>(dfbuf,ndec,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);      

        // Copy buffer to host
        checkCudaErrors(hipMemcpyAsync(cbuf[streamIdx][idm],dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost,stream));

        hipEventRecord(dmWriteEvents[streamIdx][idm], stream);

        #pragma omp task
        {
          write_to_disk_char(&(cbuf[streamIdx][idm][writeOffset*nsub/ndec]), outfile[idm], (nread-writeOffset)*nsub/ndec, dmWriteEvents[streamIdx][idm]);
        }

      } else {
        if (ndec==1) {
          checkCudaErrors(hipMemcpyAsync(cbuff[streamIdx][idm], dfbuf,sizeof(float)*msamp*mchan,hipMemcpyDeviceToHost,stream));
          
        } else {
          blocksize.x=32; blocksize.y=32; blocksize.z=1;
          gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
          decimate<<<gridsize,blocksize,0,stream>>>(dfbuf,ndec,mchan,mblock,msum,dcbuff);
          checkCudaErrors(hipMemcpyAsync(cbuff[streamIdx][idm],dcbuff,sizeof(float)*msamp*mchan/ndec,hipMemcpyDeviceToHost,stream));
        }

        hipEventRecord(dmWriteEvents[streamIdx][idm], stream);

        #pragma omp task
        {
          write_to_disk_float(&(cbuff[streamIdx][idm][writeOffset*nsub/ndec]), outfile[idm], (nread-writeOffset)*nsub/ndec, dmWriteEvents[streamIdx][idm]);
        }
      }
    }

    #pragma omp taskwait

    printf("Processed %d DMs in %.2f s\n",ndm,(float) (clock()-startclock)/CLOCKS_PER_SEC);
    timeInSeconds += (double) (nread - writeOffset) * timeOffset;
    printf("Current data processed: %02ld:%02ld:%05.2lf (%1.2lfs)\n\n", (long int) (timeInSeconds / 3600.0), (long int) ((fmod(timeInSeconds, 3600.0)) / 60.0), fmod(timeInSeconds, 60.0), timeInSeconds);
    // Exit when we pass the read length limit
    if (total_ts_read > ts_read) {
      #pragma omp cancel parallel for
    }

  }

  omp_destroy_lock(&readingLock);
  // Close files
  for (i=0;i<ndm;i++)
    fclose(outfile[i]);

  // Reader cleanup
  lofar_udp_reader_cleanup(reader);

  // Free
  free(header);
  for (i=0;i<4;i++) {
    hipFree(dudpbuf);  
  }
  free(fbuf);
  free(dm);
  free(outfile);

  if (redig) {
    for (i = 0; i < ndm; i++)
      for (j =0; j < 2; j++)
        free(cbuf[j][i]);
    hipFree(bs1);
    hipFree(bs2);
    hipFree(zavg);
    hipFree(zstd);
    hipFree(dcbuf);
  } else {
    for (i = 0; i < ndm; i++)
      for (j =0; j < 2; j++)
        free(cbuff[j][i]);
    if (ndec > 1) hipFree(dcbuff);
  }

  hipFree(dfbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(cp1p);
  hipFree(cp2p);
  hipFree(dc);
  hipFree(ddm);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  for(i = 0; i < 3; i++)
    hipStreamDestroy(streams[i]);
  for(i = 0; i < 2; i++)
    hipEventDestroy(events[i]);

  for (i = 0; i < ndm; i++)
    for (j =0; j < 2; j++)
      hipEventDestroy(dmWriteEvents[j][i]);

  return 0;
}


void inline write_to_disk_float(float* outputArray, FILE* outputFile, int nsamples, hipEvent_t waitEvent)
{
  hipEventSynchronize(waitEvent);
  fwrite(outputArray,sizeof(float),nsamples, outputFile); 
}

void inline write_to_disk_char(unsigned char* outputArray, FILE* outputFile, int nsamples, hipEvent_t waitEvent)
{
  hipEventSynchronize(waitEvent);
  fwrite(outputArray,sizeof(char),nsamples, outputFile); 
}



// Rip out sigproc's header reader. Don't have the time to spend several hours reimplementing it; all credit to Lorimer et al.
//BEGIN SIGPROC READ_HEADER.C
//
int strings_equal (char *string1, char *string2) /* includefile */
{
  if (!strcmp(string1,string2)) {
    return 1;
  } else {
    return 0;
  }
}
/* read a string from the input which looks like nchars-char[1-nchars] */
void get_string(FILE *inputfile, int *nbytes, char string[])
{
  int nchar;
  strcpy(string,"ERROR");
  if (! fread(&nchar, sizeof(int), 1, inputfile)) fprintf(stderr, "Failed to get int at %d\n", *nbytes);
  *nbytes=sizeof(int);
  if (feof(inputfile)) exit(0);
  if (nchar>80 || nchar<1) return;
  if (! fread(string, nchar, 1, inputfile)) fprintf(stderr, "Failed to get stirng at %d\n", *nbytes);
  string[nchar]='\0';
  *nbytes+=nchar;
}

/* attempt to read in the general header info from a pulsar data file */
struct header read_header(FILE *inputfile) /* includefile */
{
  char string[80], message[80];
  int nbytes,totalbytes,expecting_rawdatafile=0,expecting_source_name=0; 
  int isign=0, dummyread=0;
  struct header hdr;


  /* try to read in the first line of the header */
  get_string(inputfile,&nbytes,string);
  if (!strings_equal(string, (char *) "HEADER_START")) {
  /* the data file is not in standard format, rewind and return */
  rewind(inputfile);
  fprintf(stderr, "Unexpected input header; exiting.");
  exit(1);
  }
  /* store total number of bytes read so far */
  totalbytes=nbytes;

  /* loop over and read remaining header lines until HEADER_END reached */
  // David McKenna: We don't need all of these; ignore those values and just reference their lengths
  while (1) {
    get_string(inputfile,&nbytes,string);
    if (strings_equal(string, (char *) "HEADER_END")) break;
    totalbytes+=nbytes;
    if (strings_equal(string, (char *) "rawdatafile")) {
      expecting_rawdatafile=1;
    } else if (strings_equal(string, (char *) "source_name")) {
      expecting_source_name=1;
    } else if (strings_equal(string, (char *) "FREQUENCY_START")) {
      // pass
    } else if (strings_equal(string, (char *) "FREQUENCY_END")) {
      // pass
    } else if (strings_equal(string, (char *) "az_start")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "za_start")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "src_raj")) {
      dummyread = fread(&(hdr.src_raj),sizeof(hdr.src_raj),1,inputfile);
      totalbytes+=sizeof(hdr.src_raj);
    } else if (strings_equal(string, (char *) "src_dej")) {
      dummyread = fread(&(hdr.src_dej),sizeof(hdr.src_dej),1,inputfile);
      totalbytes+=sizeof(hdr.src_dej);
    } else if (strings_equal(string, (char *) "tstart")) {
      dummyread = fread(&(hdr.tstart),sizeof(hdr.tstart),1,inputfile);
      totalbytes+=sizeof(hdr.tstart);
    } else if (strings_equal(string, (char *) "tsamp")) {
      dummyread = fread(&(hdr.tsamp),sizeof(hdr.tsamp),1,inputfile);
      totalbytes+=sizeof(hdr.tsamp);
    } else if (strings_equal(string, (char *) "period")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "fch1")) {
      dummyread = fread(&(hdr.fch1),sizeof(hdr.fch1),1,inputfile);
      totalbytes+=sizeof(hdr.fch1);
    } else if (strings_equal(string, (char *) "fchannel")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "foff")) {
      dummyread = fread(&(hdr.foff),sizeof(hdr.foff),1,inputfile);
      totalbytes+=sizeof(hdr.foff);
    } else if (strings_equal(string, (char *) "nchans")) {
      // nsub seems to be nchans in the sigproc hdr
      dummyread = fread(&(hdr.nsub),sizeof(hdr.nsub),1,inputfile);
      totalbytes+=sizeof(hdr.nsub);
    } else if (strings_equal(string, (char *) "telescope_id")) {
      dummyread = fread(&(hdr.tel),sizeof(hdr.tel),1,inputfile);
      totalbytes+=sizeof(hdr.tel);
    } else if (strings_equal(string, (char *) "machine_id")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "data_type")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "ibeam")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbeams")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbits")) {
      dummyread = fread(&(hdr.nbit),sizeof(hdr.nbit),1,inputfile);
      totalbytes+=sizeof(hdr.nbit);
    } else if (strings_equal(string, (char *) "barycentric")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "pulsarcentric")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbins")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nsamples")) {
      /* read this one only for backwards compatibility */
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nifs")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "npuls")) {
      totalbytes+=sizeof(long int);
    } else if (strings_equal(string, (char *) "refdm")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "signed")) {
      dummyread = fread(&isign,sizeof(isign),1,inputfile);
      totalbytes+=sizeof(isign);
    } else if (expecting_rawdatafile) {
      //strcpy(hdr.rawfname,string);
      expecting_rawdatafile=0;
    } else if (expecting_source_name) {
      strcpy(hdr.source_name,string);
      expecting_source_name=0;
    } else {
      sprintf(message,"read_header (%d) - unknown parameter: %s\n", dummyread, string);
      fprintf(stderr,"ERROR: %s\n",message);
      exit(1);
    } 
    if (totalbytes != ftell(inputfile)){
      fprintf(stderr,"ERROR: Header bytes does not equal file position\n");
      fprintf(stderr,"String was: '%s'\n",string);
      fprintf(stderr,"       header: %d file: %ld\n",totalbytes,ftell(inputfile));
      exit(1);
    }


  } 

  /* add on last header string */
  totalbytes+=nbytes;

  if (totalbytes != ftell(inputfile)){
    fprintf(stderr,"ERROR: Header bytes does not equal file position\n");
    fprintf(stderr,"       header: %d file: %ld\n",totalbytes,ftell(inputfile));
    exit(1);
  }

  /* return total number of bytes read */
  return hdr;
}
// END SIGPROC READ_HEADER.c




struct header read_sigproc_header(char *fname, char *dataname, int ports)
{

  FILE *tmpf;

  tmpf = fopen(fname, "r");
  if (tmpf == NULL) {
    fprintf(stderr, "Unable to open sigproc header at %s; exiting.\n", fname);
    exit(1);
  }
  struct header hdr = read_header(tmpf);
  fclose(tmpf);



  hdr.fcen = hdr.fch1 + (hdr.foff * hdr.nsub * 0.5);
  hdr.bwchan = fabs(hdr.foff);

 return hdr;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i+nx*l]),scale);
  }
}

// Compute chirp
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c)
{
  int ibin,ichan,isub,idm,mbin,idx;
  double s,rt,t,f,fsub,fchan,bwchan,bwsub;

  // Number of channels per subband
  mbin=nbin/nchan;

  // Subband bandwidth
  bwsub=bw/nsub;

  // Channel bandwidth
  bwchan=bw/(nchan*nsub);

  // Indices of input data
  isub=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  idm=blockIdx.z*blockDim.z+threadIdx.z;

  // Keep in range
  if (isub<nsub && ichan<nchan && idm<ndm) {
    // Main constant
    s=2.0*M_PI*dm[idm]/DMCONSTANT;

    // Frequencies
    fsub=fcen-0.5*bw+bw*(float) isub/(float) nsub+0.5*bw/(float) nsub;
    fchan=fsub-0.5*bwsub+bwsub*(float) ichan/(float) nchan+0.5*bwsub/(float) nchan;
      
    // Loop over bins in channel
    for (ibin=0;ibin<mbin;ibin++) {
      // Bin frequency
      f=-0.5*bwchan+bwchan*(float) ibin/(float) mbin+0.5*bwchan/(float) mbin;
      
      // Phase delay
      rt=-f*f*s/((fchan+f)*fchan*fchan);
      
      // Taper
      t=1.0/sqrt(1.0+pow((f/(0.47*bwchan)),80));
      
      // Index
      idx=ibin+ichan*mbin+isub*mbin*nchan+idm*nsub*mbin*nchan;
      
      // Chirp
      c[idx].x=cos(rt)*t;
      c[idx].y=sin(rt)*t;
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    if (isamp >= noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(isamp-noverlap);
      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution. This is separate from the main kernel to minimise performance
// loss to branching on the GPU. On the first iteration, we want to fill
// the final non-noverlap region and final noverlap region so that they can 
// match the first noverlap region and first non-noverlap on the second
// iteration
__global__ void unpack_and_padd_first_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    if (isamp >= 2*noverlap) {
      idx1=ibin+nbin*isub+nsub*nbin*ifft;

      idx2=isub+nsub*(isamp-2*noverlap);
      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are located in the first noverlap region and first non-
// noverlap region, for continuous time series between data blocks
// 
// overlap_(timeblock)_(index)
// t = 0: overlap_0_0: nfft_0_0, nfft_0_1... nfft_0_N-1, nfft_0 N: overlap_0_1
// t = 1: nfft_0_N: overlap_0_1, nfft_1_0.... nfft_1_N-1:overlap_1_1
// t = 2 nfft_1_N-1: overlap_1_1...
// etc
__global__ void padd_next_iteration(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    if (isamp<noverlap) {
      // VVV FIX
      idx1=ibin+nbin*isub+nsub*nbin*ifft;
      idx2=isub+nsub*(isamp+nsamp-noverlap);
      cp1[idx1].x=(float) dbuf0[idx2];
      cp1[idx1].y=(float) dbuf1[idx2];
      cp2[idx1].x=(float) dbuf2[idx2];
      cp2[idx1].y=(float) dbuf3[idx2];
    }
  }
}


// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf)
{
  int64_t ibin,ichan,ifft,isub,isamp,idx1,idx2;
  
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  ifft=blockIdx.z*blockDim.z+threadIdx.z;
  if (ibin<nbin && ichan<nchan && ifft<nfft) {
    // Loop over subbands
    for (isub=0;isub<nsub;isub++) {
      // Padded array index
      //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
      idx1=ibin+ichan*nbin+(nsub-isub-1)*nbin*nchan+ifft*nbin*nchan*nsub;

      // Time index
      isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
      
      // Output array index
      idx2=(nchan-ichan-1)+isub*nchan+nsub*nchan*isamp;
      
      // Select data points from valid region
      if (ibin>=noverlap && ibin<=nbin-noverlap && isamp>=0 && isamp<nsamp)
  fbuf[idx2]=cp1[idx1].x*cp1[idx1].x+cp1[idx1].y*cp1[idx1].y+cp2[idx1].x*cp2[idx1].x+cp2[idx1].y*cp2[idx1].y;
    }
  }

  return;
}

void send_string(const char *string,FILE *file)
{
  int len, lenoff = 0;

  len=strlen(string);
  if (len > 63) {
    lenoff = len - 64;
    len = 64;
  }
  fwrite(&len,sizeof(int),1,file);
  fwrite(&(string[lenoff]),sizeof(char),len,file);

  return;
}

void send_float(const char *string,float x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(float),1,file);

  return;
}

void send_int(const char *string,int x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(int),1,file);

  return;
}

void send_double(const char *string,double x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(double),1,file);

  return;
}

double dec2sex(double x)
{
  double d,sec,min,deg;
  char sign;
  char tmp[32];

  sign=(x<0 ? '-' : ' ');
  x=3600.0*fabs(x);

  sec=fmod(x,60.0);
  x=(x-sec)/60.0;
  min=fmod(x,60.0);
  x=(x-min)/60.0;
  deg=x;

  sprintf(tmp,"%c%02d%02d%09.6lf",sign,(int) deg,(int) min,sec);
  sscanf(tmp,"%lf",&d);

  return d;
}

void write_filterbank_header(struct header h,FILE *file)
{
  //double ra,de;


  //ra=dec2sex(h.src_raj/15.0);
  //de=dec2sex(h.src_dej);
  
  send_string("HEADER_START",file);
  send_string("rawdatafile",file);
  send_string(h.rawfname[0],file);
  send_string("source_name",file);
  send_string(h.source_name,file);
  send_int("machine_id",11,file);
  send_int("telescope_id",h.tel,file);
  send_double("src_raj",h.src_raj,file);
  send_double("src_dej",h.src_dej,file);
  send_int("data_type",1,file);
  send_double("fch1",h.fch1,file);
  send_double("foff",h.foff,file);
  send_int("nchans",h.nchan,file);
  send_int("nbeams",0,file);
  send_int("ibeam",0,file);
  send_int("nbits",h.nbit,file);
  send_double("tstart",h.tstart,file);
  send_double("tsamp",h.tsamp,file);
  send_int("nifs",1,file);
  send_string("HEADER_END",file);

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2)
{
  int64_t ichan,iblock,isum,idx1,idx2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    idx1=ichan+nchan*iblock;
    bs1[idx1]=0.0;
    bs2[idx1]=0.0;
    for (isum=0;isum<nsum;isum++) {
      idx2=ichan+nchan*(isum+iblock*nsum);
      bs1[idx1]+=z[idx2];
      bs2[idx1]+=z[idx2]*z[idx2];
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd)
{
  int64_t ichan,iblock,idx1;
  double s1,s2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  if (ichan<nchan) {
    s1=0.0;
    s2=0.0;
    for (iblock=0;iblock<nblock;iblock++) {
      idx1=ichan+nchan*iblock;
      s1+=bs1[idx1];
      s2+=bs2[idx1];
    }
    zavg[ichan]=s1/(float) (nblock*nsum);
    zstd[ichan]=s2/(float) (nblock*nsum)-zavg[ichan]*zavg[ichan];
    zstd[ichan]=sqrt(zstd[ichan]);
  }

  return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1;
  float zoffset,zscale;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum++) {
      idx1=ichan+nchan*(isum+iblock*nsum);
      z[idx1]-=zoffset;
      z[idx1]*=256.0/zscale;
      cz[idx1]=(unsigned char) z[idx1];
      if (z[idx1]<0.0) cz[idx1]=0;
      if (z[idx1]>255.0) cz[idx1]=255;
    }
  }

  return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float zoffset,zscale,ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
  idx1=ichan+nchan*(isum+idec+iblock*nsum);
  ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      ztmp-=zoffset;
      ztmp*=256.0/zscale;
      cz[idx2]=(unsigned char) ztmp;
      if (ztmp<0.0) cz[idx2]=0;
      if (ztmp>255.0) cz[idx2]=255;
    }
  }

  return;
}


// Decimate the filterbank without redigitisation
__global__ void decimate(float *z,int ndec,int nchan,int nblock,int nsum,float *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
  idx1=ichan+nchan*(isum+idec+iblock*nsum);
  ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      cz[idx2]=(float) ztmp;
    }
  }

  return;
}

int reshapeRawUdp(lofar_udp_reader *reader) {

  if (lofar_udp_reader_step(reader) > 0) return 0;
  int nread = reader->meta->packetsPerIteration;

 // MODE 11

  nread *= 16;

  return nread;
}
