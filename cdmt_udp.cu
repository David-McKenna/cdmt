#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <time.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <getopt.h>
#include <limits.h>

#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

// Struct for header information
struct header {
  int nchan,nsamp,nbit=0,nsub;
  double tstart,tsamp,fch1,foff,fcen,bwchan;
  double src_raj,src_dej;
  char source_name[80];
  char *rawfname[4];
};

struct header read_sigproc_header(char *fname, char *dataname);
void get_channel_chirp(double fcen,double bw,float dm,int nchan,int nbin,int nsub,hipfftComplex *c);
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s);
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b);
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale);
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2);
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny);
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c);
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2);
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd);
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz);
void write_filterbank_header(struct header h,FILE *file);

// Usage
void usage()
{
  printf("cdmt -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -o <outputname> -s <sigproc header location> <fil prefix>\n\n");
  printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in raw udp format.\n");
  printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
  printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
  printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
  printf("-o <outputname>           Output filename [default: cdmt]\n");
  printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
  printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");
  printf("-s <bytes>       Number of bytes to skip in the filterbank before stating processing [integer, default: 0]\n");
  printf("-r <bytes>       Number of bytes to read in total from the -s offset [integer, default: length of file]\n");
  printf("-m <sigproc header location>  Sigproc header to read metadata from [default: fil prefix.sigprochdr]\n");

  return;
}

int main(int argc,char *argv[])
{
  int i,nsamp,nfft,mbin,nvalid,nchan=8,nbin=65536,noverlap=2048,nsub=20,ndm,ndec=1;
  int idm,iblock,nread,mchan,msamp,mblock,msum=1024;
  char *header,*udpbuf[4],*dudpbuf[4];
  FILE *rawfile[4],*file;
  unsigned char *cbuf,*dcbuf;
  float *fbuf,*dfbuf;
  float *bs1,*bs2,*zavg,*zstd;
  hipfftComplex *cp1,*cp2,*dc,*cp1p,*cp2p;
  hipfftHandle ftc2cf,ftc2cb;
  int idist,odist,iembed,oembed,istride,ostride;
  dim3 blocksize,gridsize;
  clock_t startclock;
  float *dm,*ddm,dm_start,dm_step;
  char fname[128],fheader[1024],*udpfname,sphdrfname[1024],obsid[128]="cdmt";
  int bytes_read;
  long int ts_read=LONG_MAX,ts_skip=0;
  long int total_ts_read=0,bytes_skip=0;
  int part=0,device=0;
  int arg=0;
  FILE **outfile;

  // Read options
  if (argc>1) {
    while ((arg=getopt(argc,argv,"d:D:ho:b:N:n:s:r:m:"))!=-1) {
      switch (arg) {
  
      case 'n':
  noverlap=atoi(optarg);
  break;

      case 'N':
  nbin=atoi(optarg);
  break;

      case 'b':
  ndec=atoi(optarg);
  break;

      case 'o':
  strcpy(obsid,optarg);
  break;

      case 'D':
  device=atoi(optarg);
  break;
  
      case 'd':
  sscanf(optarg,"%f,%f,%d",&dm_start,&dm_step,&ndm);
  break;

      case 'm':
  strcpy(sphdrfname,optarg);
  break;

      case 's':
  ts_skip=atol(optarg);
  break;
  
      case 'r':
  ts_read=atol(optarg);
  break;
      case 'h':
  usage();
  return 0;
      }
    }
  } else {
    printf("Unknown option '%c'\n", arg);
    usage();
    return 0;
  }
  udpfname=argv[optind];


  // Sanity checks to avoid voids in output filterbank
  if (nbin % 8 != 0) {
    fprintf(stderr, "ERROR: nbin must be disible by 8 (currently %d, remainder %d). Exiting.\n", nbin, nbin % 8);
    exit(1);
  }
  if ( (128 * (nbin-2*noverlap)) % 8 != 0 ) {
    fprintf(stderr, "ERROR: Valid data length must be divisible by 8 (currently %d, remainer %d). Exiting.", nbin-2*noverlap, (nbin-2*noverlap) % 8);
    exit(1);
  }

  if ((128 * (nbin-2*noverlap) / 8) % 1024 != 0) {
    fprintf(stderr, "ERROR: Interal sum cannot proceed; valid samples must be divisible by 1024 (currently %d, remainder %d).\n", (128 * (nbin-2*noverlap) / 8), (128 * (nbin-2*noverlap) / 8) % 1024);
    fprintf(stderr, "Consider using %d or %d as your forward FFT size next time. Exiting.\n", 64 * ((128 * (nbin-2*noverlap) / 8) - (128 * (nbin-2*noverlap) / 8) % 1024) / 1024 + 2 * noverlap,
                                                                                   64 * ((128 * (nbin-2*noverlap) / 8) + (1024  - (128 * (nbin-2*noverlap) / 8) % 1024)) / 1024 + 2 * noverlap);
    exit(1);
  }
  

  if (strcmp(sphdrfname, "") == 0) {
    sprintf(sphdrfname, "%s.sigprochdr", udpfname);
  }
  
  // Read sigproc header
  struct header hdr = read_sigproc_header(sphdrfname, udpfname);

  printf("====ORIGINAL HEADER INFORMATION====\n");
  printf("nsub: %d, nsamp: %d, nbit: %d, nchan %d\n", hdr.nsub, hdr.nsamp, hdr.nbit, hdr.nchan);
  printf("tstart: %lf\n", hdr.tstart);
  printf("tsamp: %.08lf\n", hdr.tsamp);
  printf("fch1: %lf\n", hdr.fch1);
  printf("foff: %lf\n", hdr.foff);
  printf("fcen: %lf\n", hdr.fcen);
  printf("bwchan: %lf\n", hdr.bwchan);
  printf("src_raj: %lf\n", hdr.src_raj);
  printf("src_dej: %lf\n", hdr.src_dej);
  printf("source: %s\n", hdr.source_name);
  printf("====ORIGINAL HEADER INFORMATION====\n");

  // Handle skip flag
  if (ts_skip > 0) {
    // If it's initialised by default...
    if (hdr.nbit == 0)
      hdr.nbit = 8;
    bytes_skip = (long int) (ts_skip * (float) hdr.nsub * (float) hdr.nbit / 8.0);
    // Account for the difference in time in the new header if we skip bytes    // tstart = MJD, tsamp = seconds, 1 byte = 8 bits = 1 sample per file by default
    hdr.tstart += (double) ts_skip * hdr.tsamp / 86400.0;
  }

  // Read the number of subbands
  nsub=hdr.nsub;

  // Adjust header for filterbank format
  hdr.tsamp*=nchan*ndec;
  hdr.nchan=nsub*nchan;
  hdr.nbit=8;
  hdr.fch1=hdr.fcen+0.5*hdr.nsub*hdr.bwchan-0.5*hdr.bwchan/nchan;
  hdr.foff=-fabs(hdr.bwchan/nchan);


  printf("====NEW HEADER INFORMATION====\n");
  printf("nsub: %d, nsamp: %d, nbit: %d, nchan %d\n", hdr.nsub, hdr.nsamp, hdr.nbit, hdr.nchan);
  printf("tstart: %lf\n", hdr.tstart);
  printf("tsamp: %.08lf\n", hdr.tsamp);
  printf("fch1: %lf\n", hdr.fch1);
  printf("foff: %lf\n", hdr.foff);
  printf("fcen: %lf\n", hdr.fcen);
  printf("bwchan: %lf\n", hdr.bwchan);
  printf("src_raj: %lf\n", hdr.src_raj);
  printf("src_dej: %lf\n", hdr.src_dej);
  printf("source: %s\n", hdr.source_name);
  printf("====NEW HEADER INFORMATION====\n");

  // Data size
  nvalid=nbin-2*noverlap;
  nsamp=128*nvalid;
  nfft=(int) ceil(nsamp/(float) nvalid);
  mbin=nbin/nchan; // nbin must be evenly divisible by 8
  mchan=nsub*nchan;
  msamp=nsamp/nchan; // 128 * nvalid must be divisble by 8
  mblock=msamp/msum; // 128 * nvalid / 8 must be disible by 1024

  printf("nbin: %d nfft: %d nsub: %d mbin: %d nchan: %d nsamp: %d nvalid: %d\n",nbin,nfft,nsub,mbin,nchan,nsamp,nvalid);
  printf("msamp: %d mblock: %d mchan: %d\n", msamp, mblock, mchan);

  // Set device
  checkCudaErrors(hipSetDevice(device));

  // DMcK: cuFFT docs say it's best practice to plan before allocating memory
  // cuda-memcheck fails initialisation before this block is run?
  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nfft*nsub));
  hipDeviceSynchronize();

  // Generate FFT plan (batch in-place backward FFT)
  idist=mbin;  odist=mbin;  iembed=mbin;  oembed=mbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cb,1,&mbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nchan*nfft*nsub));
  hipDeviceSynchronize();

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,  (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2,  (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp1p, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));
  checkCudaErrors(hipMalloc((void **) &cp2p, (size_t) sizeof(hipfftComplex)*nbin*nfft*nsub));

  // Allocate device memory for chirp
  checkCudaErrors(hipMalloc((void **) &dc, (size_t) sizeof(hipfftComplex)*nbin*nsub*ndm));

  // Allocate device memory for block sums
  checkCudaErrors(hipMalloc((void **) &bs1, (size_t) sizeof(float)*mblock*mchan));
  checkCudaErrors(hipMalloc((void **) &bs2, (size_t) sizeof(float)*mblock*mchan));

  // Allocate device memory for channel averages and standard deviations
  checkCudaErrors(hipMalloc((void **) &zavg, (size_t) sizeof(float)*mchan));
  checkCudaErrors(hipMalloc((void **) &zstd, (size_t) sizeof(float)*mchan));

  // Allocate memory for redigitized output and header
  header=(char *) malloc(sizeof(char)*HEADERSIZE);
  for (i=0;i<4;i++) {
    udpbuf[i]=(char *) malloc(sizeof(char)*nsamp*nsub);
    checkCudaErrors(hipMalloc((void **) &dudpbuf[i], (size_t) sizeof(char)*nsamp*nsub));
  }

  // Allocate output buffers
  fbuf=(float *) malloc(sizeof(float)*nsamp*nsub);
  checkCudaErrors(hipMalloc((void **) &dfbuf, (size_t) sizeof(float)*nsamp*nsub));
  cbuf=(unsigned char *) malloc(sizeof(unsigned char)*msamp*mchan/ndec);
  checkCudaErrors(hipMalloc((void **) &dcbuf, (size_t) sizeof(unsigned char)*msamp*mchan/ndec));

  // Allocate DMs and copy to device
  dm=(float *) malloc(sizeof(float)*ndm);
  for (idm=0;idm<ndm;idm++)
    dm[idm]=dm_start+(float) idm*dm_step;
  checkCudaErrors(hipMalloc((void **) &ddm, (size_t) sizeof(float)*ndm));
  checkCudaErrors(hipMemcpy(ddm,dm,sizeof(float)*ndm,hipMemcpyHostToDevice));

  // Allow memory alloation/copy actions to finish before processing
  hipDeviceSynchronize();

  // Compute chirp
  blocksize.x=32; blocksize.y=32; blocksize.z=1;
  gridsize.x=nsub/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=ndm/blocksize.z+1;
  compute_chirp<<<gridsize,blocksize>>>(hdr.fcen,nsub*hdr.bwchan,ddm,nchan,nbin,nsub,ndm,dc);

  // Write temporary filterbank header
  file=fopen("/tmp/header.fil","w");
  if (file == NULL) {
    fprintf(stderr, "ERROR: Unable to open /tmp/header.fil to write temporary header; exiting.\n");
    exit(1);
  }
  write_filterbank_header(hdr,file);
  fclose(file);
  file=fopen("/tmp/header.fil","r");
  if (file == NULL) {
    fprintf(stderr, "ERROR: Unable to re-open /tmp/header.fil to read temporary header length; exiting.\n");
    exit(1);
  }
  bytes_read=fread(fheader,sizeof(char),1024,file);
  fclose(file);
  
  // Format file names and open
  outfile=(FILE **) malloc(sizeof(FILE *)*ndm);
  for (idm=0;idm<ndm;idm++) {
    sprintf(fname,"%s_cDM%06.2f_P%03d.fil",obsid,dm[idm],part);

    outfile[idm]=fopen(fname,"w");
    if (outfile[idm] == NULL) {
      fprintf(stderr, "Unable to open output file %s, exiting.\n", fname);
      exit(1);
    }
  }
  
  // Write headers
  for (idm=0;idm<ndm;idm++) {
    // Send header
    fwrite(fheader,sizeof(char),bytes_read,outfile[idm]);
  }

  // Read files
  for (i=0;i<4;i++) {
    rawfile[i]=fopen(hdr.rawfname[i],"r");
    if (bytes_skip > 0)
      fseek(rawfile[i],bytes_skip,SEEK_SET);
  }

  // Loop over input file contents
  for (iblock=0;;iblock++) {
    // Read block
    startclock=clock();
    for (i=0;i<4;i++)
      nread=fread(udpbuf[i],sizeof(char),nsamp*nsub,rawfile[i])/nsub;
    if (nread==0) {
      printf("No data read from last file; assuming EOF, finishng up.\n");
      break;
    }

    // Count up the total bytes read
    total_ts_read += nread * nsub;

    printf("Block: %d: Read %ld MB in %.2f s\n",iblock,sizeof(char)*nread*nsub*4/(1<<20),(float) (clock()-startclock)/CLOCKS_PER_SEC);

    // Copy buffers to device
    startclock=clock();
    for (i=0;i<4;i++)
      checkCudaErrors(hipMemcpy(dudpbuf[i],udpbuf[i],sizeof(char)*nread*nsub,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=nsub/blocksize.z+1;
    unpack_and_padd<<<gridsize,blocksize>>>(dudpbuf[0],dudpbuf[1],dudpbuf[2],dudpbuf[3],nread,nbin,nfft,nsub,noverlap,cp1p,cp2p);

    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1p,(hipfftComplex *) cp1p,HIPFFT_FORWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2p,(hipfftComplex *) cp2p,HIPFFT_FORWARD));

    // Swap spectrum halves for large FFTs
    blocksize.x=32; blocksize.y=32; blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1; gridsize.y=nfft*nsub/blocksize.y+1; gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1p,cp2p,nbin,nfft*nsub);

    // Loop over dms
    for (idm=0;idm<ndm;idm++) {

      // Perform complex multiplication of FFT'ed data with chirp
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=nbin*nsub/blocksize.x+1; gridsize.y=nfft/blocksize.y+1; gridsize.z=1;
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1p,dc,cp1,nbin*nsub,nfft,idm,1.0/(float) nbin);
      PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2p,dc,cp2,nbin*nsub,nfft,idm,1.0/(float) nbin);
      
      // Swap spectrum halves for small FFTs
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan*nfft*nsub/blocksize.y+1; gridsize.z=1;
      swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan*nfft*nsub);
      
      // Perform FFTs
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
      checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
      
      // Detect data
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mbin/blocksize.x+1; gridsize.y=nchan/blocksize.y+1; gridsize.z=nfft/blocksize.z+1;
      transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mbin,nchan,nfft,nsub,noverlap/nchan,nread/nchan,dfbuf);
      
      // Compute block sums for redigitization
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      compute_block_sums<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,bs1,bs2);
      
      // Compute channel stats
      blocksize.x=32; blocksize.y=1; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=1; gridsize.z=1;
      compute_channel_statistics<<<gridsize,blocksize>>>(mchan,mblock,msum,bs1,bs2,zavg,zstd);

      // Redigitize data to 8bits
      blocksize.x=32; blocksize.y=32; blocksize.z=1;
      gridsize.x=mchan/blocksize.x+1; gridsize.y=mblock/blocksize.y+1; gridsize.z=1;
      if (ndec==1)
  redigitize<<<gridsize,blocksize>>>(dfbuf,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);
      else
  decimate_and_redigitize<<<gridsize,blocksize>>>(dfbuf,ndec,mchan,mblock,msum,zavg,zstd,3.0,5.0,dcbuf);      

      // Copy buffer to host
      checkCudaErrors(hipMemcpy(cbuf,dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost));

      // Write buffer
      fwrite(cbuf,sizeof(char),nread*nsub/ndec,outfile[idm]);
    }
    printf("Processed %d DMs in %.2f s\n",ndm,(float) (clock()-startclock)/CLOCKS_PER_SEC);

    // Exit when we pass the read length limit
    if (total_ts_read > ts_read)
      break;
  }

  // Close files
  for (i=0;i<ndm;i++)
    fclose(outfile[i]);

  // Close files
  for (i=0;i<4;i++)
    fclose(rawfile[i]);

  // Free
  free(header);
  for (i=0;i<4;i++) {
    free(udpbuf[i]);
    hipFree(dudpbuf);
    free(hdr.rawfname[i]);
  }
  free(fbuf);
  free(dm);
  free(cbuf);
  free(outfile);

  hipFree(dfbuf);
  hipFree(dcbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(cp1p);
  hipFree(cp2p);
  hipFree(dc);
  hipFree(bs1);
  hipFree(bs2);
  hipFree(zavg);
  hipFree(zstd);
  hipFree(ddm);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}



// Rip out sigproc's header reader. Don't have the time to spend several hours reimplementing it; all credit to Lorimer et al.
//BEGIN SIGPROC READ_HEADER.C
//
int strings_equal (char *string1, char *string2) /* includefile */
{
  if (!strcmp(string1,string2)) {
    return 1;
  } else {
    return 0;
  }
}
/* read a string from the input which looks like nchars-char[1-nchars] */
void get_string(FILE *inputfile, int *nbytes, char string[])
{
  int nchar;
  strcpy(string,"ERROR");
  fread(&nchar, sizeof(int), 1, inputfile);
  *nbytes=sizeof(int);
  if (feof(inputfile)) exit(0);
  if (nchar>80 || nchar<1) return;
  fread(string, nchar, 1, inputfile);
  string[nchar]='\0';
  *nbytes+=nchar;
}

/* attempt to read in the general header info from a pulsar data file */
struct header read_header(FILE *inputfile) /* includefile */
{
  char string[80], message[80];
  int nbytes,totalbytes,expecting_rawdatafile=0,expecting_source_name=0; 
  int isign=0;
  struct header hdr;


  /* try to read in the first line of the header */
  get_string(inputfile,&nbytes,string);
  if (!strings_equal(string, (char *) "HEADER_START")) {
  /* the data file is not in standard format, rewind and return */
  rewind(inputfile);
  fprintf(stderr, "Unexpected input header; exiting.");
  exit(1);
  }
  /* store total number of bytes read so far */
  totalbytes=nbytes;

  /* loop over and read remaining header lines until HEADER_END reached */
  // David McKenna: We don't need all of these; ignore those values and just reference their lengths
  while (1) {
    get_string(inputfile,&nbytes,string);
    if (strings_equal(string, (char *) "HEADER_END")) break;
    totalbytes+=nbytes;
    if (strings_equal(string, (char *) "rawdatafile")) {
      expecting_rawdatafile=1;
    } else if (strings_equal(string, (char *) "source_name")) {
      expecting_source_name=1;
    } else if (strings_equal(string, (char *) "FREQUENCY_START")) {
      // pass
    } else if (strings_equal(string, (char *) "FREQUENCY_END")) {
      // pass
    } else if (strings_equal(string, (char *) "az_start")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "za_start")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "src_raj")) {
      fread(&(hdr.src_raj),sizeof(hdr.src_raj),1,inputfile);
      totalbytes+=sizeof(hdr.src_raj);
    } else if (strings_equal(string, (char *) "src_dej")) {
      fread(&(hdr.src_dej),sizeof(hdr.src_dej),1,inputfile);
      totalbytes+=sizeof(hdr.src_dej);
    } else if (strings_equal(string, (char *) "tstart")) {
      fread(&(hdr.tstart),sizeof(hdr.tstart),1,inputfile);
      totalbytes+=sizeof(hdr.tstart);
    } else if (strings_equal(string, (char *) "tsamp")) {
      fread(&(hdr.tsamp),sizeof(hdr.tsamp),1,inputfile);
      totalbytes+=sizeof(hdr.tsamp);
    } else if (strings_equal(string, (char *) "period")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "fch1")) {
      fread(&(hdr.fch1),sizeof(hdr.fch1),1,inputfile);
      totalbytes+=sizeof(hdr.fch1);
    } else if (strings_equal(string, (char *) "fchannel")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "foff")) {
      fread(&(hdr.foff),sizeof(hdr.foff),1,inputfile);
      totalbytes+=sizeof(hdr.foff);
    } else if (strings_equal(string, (char *) "nchans")) {
      // nsub seems to be nchans in the sigproc hdr
      fread(&(hdr.nsub),sizeof(hdr.nsub),1,inputfile);
      totalbytes+=sizeof(hdr.nsub);
    } else if (strings_equal(string, (char *) "telescope_id")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "machine_id")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "data_type")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "ibeam")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbeams")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbits")) {
      fread(&(hdr.nbit),sizeof(hdr.nbit),1,inputfile);
      totalbytes+=sizeof(hdr.nbit);
    } else if (strings_equal(string, (char *) "barycentric")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "pulsarcentric")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nbins")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nsamples")) {
      /* read this one only for backwards compatibility */
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "nifs")) {
      fseek(inputfile, sizeof(int), SEEK_CUR);
      totalbytes+=sizeof(int);
    } else if (strings_equal(string, (char *) "npuls")) {
      totalbytes+=sizeof(long int);
    } else if (strings_equal(string, (char *) "refdm")) {
      fseek(inputfile, sizeof(double), SEEK_CUR);
      totalbytes+=sizeof(double);
    } else if (strings_equal(string, (char *) "signed")) {
      fread(&isign,sizeof(isign),1,inputfile);
      totalbytes+=sizeof(isign);
    } else if (expecting_rawdatafile) {
      //strcpy(hdr.rawfname,string);
      expecting_rawdatafile=0;
    } else if (expecting_source_name) {
      strcpy(hdr.source_name,string);
      expecting_source_name=0;
    } else {
      sprintf(message,"read_header - unknown parameter: %s\n",string);
      fprintf(stderr,"ERROR: %s\n",message);
      exit(1);
    } 
    if (totalbytes != ftell(inputfile)){
      fprintf(stderr,"ERROR: Header bytes does not equal file position\n");
      fprintf(stderr,"String was: '%s'\n",string);
      fprintf(stderr,"       header: %d file: %ld\n",totalbytes,ftell(inputfile));
      exit(1);
    }


  } 

  /* add on last header string */
  totalbytes+=nbytes;

  if (totalbytes != ftell(inputfile)){
    fprintf(stderr,"ERROR: Header bytes does not equal file position\n");
    fprintf(stderr,"       header: %d file: %ld\n",totalbytes,ftell(inputfile));
    exit(1);
  }

  /* return total number of bytes read */
  return hdr;
}
// END SIGPROC READ_HEADER.c




struct header read_sigproc_header(char *fname, char *dataname)
{

  char ftest[2048];
  int i;
  FILE *tmpf;

  tmpf = fopen(fname, "r");
  if (tmpf == NULL) {
    fprintf(stderr, "Unable to open sigproc header at %s; exiting.\n", fname);
    exit(1);
  }
  struct header hdr = read_header(tmpf);
  fclose(tmpf);


  // Check files
  for (i=0;i<4;i++) {
    // Format file name
    sprintf(ftest,"%s_S%d.rawfil",dataname,i);
    // Try to open
    if ((tmpf=fopen(ftest,"r"))!=NULL) {
      fclose(tmpf);
    } else {
      fprintf(stderr,"Raw file %s not found\n",ftest);
      exit (-1);
    }
    hdr.rawfname[i]=(char *) malloc(sizeof(char) * strlen(dataname) + sizeof(char)*(9));
    strcpy(hdr.rawfname[i],ftest);
  }

  tmpf = fopen(hdr.rawfname[0], "r");
  fseek(tmpf, 0, SEEK_END);
  long int charSize = ftell(tmpf);
  fclose(tmpf);



  hdr.fcen = hdr.fch1 + (hdr.foff * hdr.nsub * 0.5);
  hdr.bwchan = fabs(hdr.foff);

  hdr.nsamp = (int) (charSize / hdr.nsub / ((float) (hdr.nbit) / (float) 8));

  return hdr;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,int l,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i+nx*l]),scale);
  }
}

// Compute chirp
__global__ void compute_chirp(double fcen,double bw,float *dm,int nchan,int nbin,int nsub,int ndm,hipfftComplex *c)
{
  int ibin,ichan,isub,idm,mbin,idx;
  double s,rt,t,f,fsub,fchan,bwchan,bwsub;

  // Number of channels per subband
  mbin=nbin/nchan;

  // Subband bandwidth
  bwsub=bw/nsub;

  // Channel bandwidth
  bwchan=bw/(nchan*nsub);

  // Indices of input data
  isub=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  idm=blockIdx.z*blockDim.z+threadIdx.z;

  // Keep in range
  if (isub<nsub && ichan<nchan && idm<ndm) {
    // Main constant
    s=2.0*M_PI*dm[idm]/DMCONSTANT;

    // Frequencies
    fsub=fcen-0.5*bw+bw*(float) isub/(float) nsub+0.5*bw/(float) nsub;
    fchan=fsub-0.5*bwsub+bwsub*(float) ichan/(float) nchan+0.5*bwsub/(float) nchan;
      
    // Loop over bins in channel
    for (ibin=0;ibin<mbin;ibin++) {
      // Bin frequency
      f=-0.5*bwchan+bwchan*(float) ibin/(float) mbin+0.5*bwchan/(float) mbin;
      
      // Phase delay
      rt=-f*f*s/((fchan+f)*fchan*fchan);
      
      // Taper
      t=1.0/sqrt(1.0+pow((f/(0.47*bwchan)),80));
      
      // Index
      idx=ibin+ichan*mbin+isub*mbin*nchan+idm*nsub*mbin*nchan;
      
      // Chirp
      c[idx].x=cos(rt)*t;
      c[idx].y=sin(rt)*t;
    }
  }

  return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char *dbuf0,char *dbuf1,char *dbuf2,char *dbuf3,int nsamp,int nbin,int nfft,int nsub,int noverlap,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t ibin,ifft,isamp,isub,idx1,idx2;

  // Indices of input data
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ifft=blockIdx.y*blockDim.y+threadIdx.y;
  isub=blockIdx.z*blockDim.z+threadIdx.z;

  // Only compute valid threads
  if (ibin<nbin && ifft<nfft && isub<nsub) {
    idx1=ibin+nbin*isub+nsub*nbin*ifft;
    isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
    idx2=isub+nsub*isamp;
    if (isamp<0) {
      idx2 *= -1;
    } else if (isamp>=nsamp) {
      idx2 -= 2 * (isamp - nsamp + 1) * nsub;
    } 

    cp1[idx1].x=(float) dbuf0[idx2];
    cp1[idx1].y=(float) dbuf1[idx2];
    cp2[idx1].x=(float) dbuf2[idx2];
    cp2[idx1].y=(float) dbuf3[idx2];
  }

  return;
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nbin,int nchan,int nfft,int nsub,int noverlap,int nsamp,float *fbuf)
{
  int64_t ibin,ichan,ifft,isub,isamp,idx1,idx2;
  
  ibin=blockIdx.x*blockDim.x+threadIdx.x;
  ichan=blockIdx.y*blockDim.y+threadIdx.y;
  ifft=blockIdx.z*blockDim.z+threadIdx.z;
  if (ibin<nbin && ichan<nchan && ifft<nfft) {
    // Loop over subbands
    for (isub=0;isub<nsub;isub++) {
      // Padded array index
      //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
      idx1=ibin+ichan*nbin+(nsub-isub-1)*nbin*nchan+ifft*nbin*nchan*nsub;

      // Time index
      isamp=ibin+(nbin-2*noverlap)*ifft-noverlap;
      
      // Output array index
      idx2=(nchan-ichan-1)+isub*nchan+nsub*nchan*isamp;
      
      // Select data points from valid region
      if (ibin>=noverlap && ibin<=nbin-noverlap && isamp>=0 && isamp<nsamp)
  fbuf[idx2]=cp1[idx1].x*cp1[idx1].x+cp1[idx1].y*cp1[idx1].y+cp2[idx1].x*cp2[idx1].x+cp2[idx1].y*cp2[idx1].y;
    }
  }

  return;
}

void send_string(const char *string,FILE *file)
{
  int len;

  len=strlen(string);
  fwrite(&len,sizeof(int),1,file);
  fwrite(string,sizeof(char),len,file);

  return;
}

void send_float(const char *string,float x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(float),1,file);

  return;
}

void send_int(const char *string,int x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(int),1,file);

  return;
}

void send_double(const char *string,double x,FILE *file)
{
  send_string(string,file);
  fwrite(&x,sizeof(double),1,file);

  return;
}

double dec2sex(double x)
{
  double d,sec,min,deg;
  char sign;
  char tmp[32];

  sign=(x<0 ? '-' : ' ');
  x=3600.0*fabs(x);

  sec=fmod(x,60.0);
  x=(x-sec)/60.0;
  min=fmod(x,60.0);
  x=(x-min)/60.0;
  deg=x;

  sprintf(tmp,"%c%02d%02d%09.6lf",sign,(int) deg,(int) min,sec);
  sscanf(tmp,"%lf",&d);

  return d;
}

void write_filterbank_header(struct header h,FILE *file)
{
  //double ra,de;


  //ra=dec2sex(h.src_raj/15.0);
  //de=dec2sex(h.src_dej);
  
  send_string("HEADER_START",file);
  send_string("rawdatafile",file);
  send_string(h.rawfname[0],file);
  send_string("source_name",file);
  send_string(h.source_name,file);
  send_int("machine_id",11,file);
  send_int("telescope_id",11,file);
  send_double("src_raj",h.src_raj,file);
  send_double("src_dej",h.src_dej,file);
  send_int("data_type",1,file);
  send_double("fch1",h.fch1,file);
  send_double("foff",h.foff,file);
  send_int("nchans",h.nchan,file);
  send_int("nbeams",0,file);
  send_int("ibeam",0,file);
  send_int("nbits",h.nbit,file);
  send_double("tstart",h.tstart,file);
  send_double("tsamp",h.tsamp,file);
  send_int("nifs",1,file);
  send_string("HEADER_END",file);

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float *z,int nchan,int nblock,int nsum,float *bs1,float *bs2)
{
  int64_t ichan,iblock,isum,idx1,idx2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    idx1=ichan+nchan*iblock;
    bs1[idx1]=0.0;
    bs2[idx1]=0.0;
    for (isum=0;isum<nsum;isum++) {
      idx2=ichan+nchan*(isum+iblock*nsum);
      bs1[idx1]+=z[idx2];
      bs2[idx1]+=z[idx2]*z[idx2];
    }
  }

  return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan,int nblock,int nsum,float *bs1,float *bs2,float *zavg,float *zstd)
{
  int64_t ichan,iblock,idx1;
  double s1,s2;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  if (ichan<nchan) {
    s1=0.0;
    s2=0.0;
    for (iblock=0;iblock<nblock;iblock++) {
      idx1=ichan+nchan*iblock;
      s1+=bs1[idx1];
      s2+=bs2[idx1];
    }
    zavg[ichan]=s1/(float) (nblock*nsum);
    zstd[ichan]=s2/(float) (nblock*nsum)-zavg[ichan]*zavg[ichan];
    zstd[ichan]=sqrt(zstd[ichan]);
  }

  return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float *z,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1;
  float zoffset,zscale;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum++) {
      idx1=ichan+nchan*(isum+iblock*nsum);
      z[idx1]-=zoffset;
      z[idx1]*=256.0/zscale;
      cz[idx1]=(unsigned char) z[idx1];
      if (z[idx1]<0.0) cz[idx1]=0;
      if (z[idx1]>255.0) cz[idx1]=255;
    }
  }

  return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float *z,int ndec,int nchan,int nblock,int nsum,float *zavg,float *zstd,float zmin,float zmax,unsigned char *cz)
{
  int64_t ichan,iblock,isum,idx1,idx2,idec;
  float zoffset,zscale,ztmp;

  ichan=blockIdx.x*blockDim.x+threadIdx.x;
  iblock=blockIdx.y*blockDim.y+threadIdx.y;
  if (ichan<nchan && iblock<nblock) {
    zoffset=zavg[ichan]-zmin*zstd[ichan];
    zscale=(zmin+zmax)*zstd[ichan];

    for (isum=0;isum<nsum;isum+=ndec) {
      idx2=ichan+nchan*(isum/ndec+iblock*nsum/ndec);
      for (idec=0,ztmp=0.0;idec<ndec;idec++) {
  idx1=ichan+nchan*(isum+idec+iblock*nsum);
  ztmp+=z[idx1];
      }
      ztmp/=(float) ndec;
      ztmp-=zoffset;
      ztmp*=256.0/zscale;
      cz[idx2]=(unsigned char) ztmp;
      if (ztmp<0.0) cz[idx2]=0;
      if (ztmp>255.0) cz[idx2]=255;
    }
  }

  return;
}
