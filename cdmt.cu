#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

// Compute chirp                                                                
void get_channel_chirp(double f0,double df,float dm,int nchan,int nbin,hipfftComplex *c)
{
  int i,k,l,m;
  float s,rt,t,f,fc0,dfc;

  // Main constant
  s=2.0*M_PI*dm/DMCONSTANT;

  // Number of channels per subband
  m=nbin/nchan;

  dfc=df/nchan;

  // Loop over subbands
  for (k=0;k<nchan;k++) {
    fc0=f0-0.5*df+df*(float) k/(float) nchan+0.5*df/(float) nchan;
    for (i=0;i<m;i++) {
      f=-0.5*dfc+dfc*(float) i/(float) (m-1);

      rt=-f*f*s/((fc0+f)*fc0*fc0);
      t=1.0/sqrt(1.0+pow((f/(0.47*dfc)),80));

      l=i+k*m;
      
      c[l].x=cos(rt)*t;
      c[l].y=sin(rt)*t;
    }
  }

  return;
}

static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)                               
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i]),scale);
  }
}

__global__ void unpack_and_padd(char *dbuf,int n,int nx,int ny,int i0,int m,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t i,j,k,l;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx && j<ny) {
    k=i+nx*j;
    l=i+m*j-i0;
    if (l<0 || l>=n) {
      cp1[k].x=0.0;
      cp1[k].y=0.0;
      cp2[k].x=0.0;
      cp2[k].y=0.0;
    } else {
      cp1[k].x=(float) dbuf[4*l];
      cp1[k].y=(float) dbuf[4*l+1];
      cp2[k].x=(float) dbuf[4*l+2];
      cp2[k].y=(float) dbuf[4*l+3];
    }
  }

  return;
}

__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny,int nz,int i0,int i1,int n,float *fbuf)
{
  int64_t i,j,k,l,m,ii,jj;
  
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  k=blockIdx.z*blockDim.z+threadIdx.z;
  if (i<nx && j<ny && k<nz) {
    m=i+nx*(j+ny*k);

    // Time index
    ii=i+(nx-i0-i1)*k-i0;

    // Frequency index
    jj=ny-j-1;

    // Array index
    l=jj+ny*ii;

    if (i>=i0 && i<=nx-i1 && ii>=0 && ii<n)
      fbuf[l]=sqrt(cp1[m].x*cp1[m].x+cp1[m].y*cp1[m].y+cp2[m].x*cp2[m].x+cp2[m].y*cp2[m].y);
  }

  return;
}

int main(int argc,char *argv[])
{
  int nsamp,nz,mx,my,mz,m,nchan=8,nbin=65536,noverlap=2048;
  int iblock,nread;
  char *header,*hbuf,*dbuf;
  FILE *file,*ofile;
  float *fbuf,*dfbuf;
  hipfftComplex *cp1,*cp2,*dc,*c;
  hipfftHandle ftc2cf,ftc2cb;
  int idist,odist,iembed,oembed,istride,ostride;
  dim3 blocksize,gridsize;

  c=(hipfftComplex *) malloc(sizeof(hipfftComplex)*nbin);

  // Compute chirp
  //get_channel_chirp(119.7265625,0.1953125,39.659298,nchan,nbin,c);
  get_channel_chirp(135.0,0.1953125,100.0,nchan,nbin,c);

  nsamp=195312.5;
  nsamp*=600;
  nsamp=491520*20;

  // Data size
  m=nbin-2*noverlap;
  nz=(int) ceil(nsamp/(float) m);
  my=nchan;
  mx=nbin/my;
  mz=nz;
  printf("%dx%d %dx%dx%d %d\n",nbin,nz,mx,my,mz,m);

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,sizeof(hipfftComplex)*nbin*nz));
  checkCudaErrors(hipMalloc((void **) &cp2,sizeof(hipfftComplex)*nbin*nz));

  // Allocate device memory for chirp                                                                
  checkCudaErrors(hipMalloc((void **) &dc,sizeof(hipfftComplex)*nbin));

  // Allocate memory for redigitized output and header
  header=(char *) malloc(sizeof(char)*HEADERSIZE);
  hbuf=(char *) malloc(sizeof(char)*4*nsamp);
  checkCudaErrors(hipMalloc((void **) &dbuf,sizeof(char)*4*nsamp));

  // Allocate output buffers
  fbuf=(float *) malloc(sizeof(float)*nsamp);
  checkCudaErrors(hipMalloc((void **) &dfbuf,sizeof(float)*nsamp));

  // Generate FFT plan (batch in-place forward FFT)
  idist=nbin;  odist=nbin;  iembed=nbin;  oembed=nbin;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cf,1,&nbin,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,nz));

  // Generate FFT plan (batch in-place backward FFT)
  idist=mx;  odist=mx;  iembed=mx;  oembed=mx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cb,1,&mx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,my*mz));

  // Copy chirp to device                                                                            
  checkCudaErrors(hipMemcpy(dc,c,sizeof(hipfftComplex)*nbin,hipMemcpyHostToDevice));

  // Read fil file header and dump in output file
  file=fopen("header.fil","r");
  fread(header,sizeof(char),351,file);
  fclose(file);
  ofile=fopen("test.fil","w");
  fwrite(header,sizeof(char),351,ofile);

  // Read file and buffer
  file=fopen("test.dada","r");
  fread(header,sizeof(char),HEADERSIZE,file);

  // Loop over input file contents
  for (iblock=0;;iblock++) {
    nread=fread(hbuf,sizeof(char),4*nsamp,file)/4;
    printf("%d %d %d\n",iblock,nread,4*nsamp);
    if (nread==0)
      break;

    // Copy buffer to device
    checkCudaErrors(hipMemcpy(dbuf,hbuf,sizeof(char)*4*nread,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1;
    gridsize.y=nz/blocksize.y+1;
    gridsize.z=1;
    unpack_and_padd<<<gridsize,blocksize>>>(dbuf,nread,nbin,nz,noverlap,m,cp1,cp2);
    
    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_FORWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_FORWARD));
    
    // Swap spectrum halves for large FFTs
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx*my/blocksize.x+1;
    gridsize.y=mz/blocksize.y+1;
    gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mx*my,mz);
    
    // Perform complex multiplication of FFT'ed data with chirp (in place)                             
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=nbin/blocksize.x+1;
    gridsize.y=nz/blocksize.y+1;
    gridsize.z=1;
    PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1,dc,cp1,nbin,nz,1.0/(float) nbin);
    PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2,dc,cp2,nbin,nz,1.0/(float) nbin);
    
    // Swap spectrum halves for small FFTs
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx/blocksize.x+1;
    gridsize.y=my*mz/blocksize.y+1;
    gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mx,my*mz);
    
    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
    
    // Detect data
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx/blocksize.x+1;
    gridsize.y=my/blocksize.y+1;
    gridsize.z=mz/blocksize.z+1;
    transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mx,my,mz,noverlap/my,noverlap/my,nread/my,dfbuf);
    
    // Copy buffer to host
    checkCudaErrors(hipMemcpy(fbuf,dfbuf,sizeof(float)*nread,hipMemcpyDeviceToHost));
    
    // Write buffer
    fwrite(fbuf,sizeof(float),nread,ofile);

  }

  // Close files
  fclose(ofile);
  fclose(file);

  // Free
  free(header);
  free(hbuf);
  free(fbuf);
  free(c);
  hipFree(dbuf);
  hipFree(dfbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(dc);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}
