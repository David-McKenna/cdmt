#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

struct chirp {
  int nbin,nd1,nd2,nd;
  hipfftComplex *c;
};

// Compute chirp                                                                
struct chirp get_chirp(double f0,double df,float dm)
{
  int i,nexp;
  float tdm;
  float s,rt,t,f,td1,td2;
  struct chirp c;

  // Compute dispersion sweep
  td1=dm*(pow(f0,-2)-pow(f0+0.5*df,-2))/DMCONSTANT;
  td2=dm*(pow(f0-0.5*df,-2)-pow(f0,-2))/DMCONSTANT;
  c.nd1=(int) floor(td1*df);
  c.nd2=(int) floor(td2*df);
  tdm=dm*(pow(f0-0.5*df,-2)-pow(f0+0.5*df,-2))/DMCONSTANT;
  c.nd=(int) floor(tdm*df);
  //  c.nd=c.nd1+c.nd2;                                                         
  nexp=(int) ceil(log(c.nd)/log(2.0))+1;
  //////////////////// HARDCODED 512k bins //////////////////////////           
  nexp=19;                                                                      
  //  c.nd1=16384;                                                                  
  //  c.nd2=16384;                                                                  
  c.nd1=65536;
  c.nd2=65536;
  c.nd=c.nd1+c.nd2;                                                             
  c.nbin=(int) pow(2.0,nexp);

  s=2.0*M_PI*dm/DMCONSTANT;
  printf("Dispersion sweep: %f us, %d bins\n%d (%d+%d) bins discarded per FFT\n\
",tdm,c.nbin,c.nd,c.nd1,c.nd2);

  // Allocate                                                                   
  c.c=(hipfftComplex *) malloc(sizeof(hipfftComplex)*c.nbin);

  // Compute chirp                                                              
  for (i=0;i<c.nbin;i++) {
    //    if (i<c.nbin/2)
    //      j=i+c.nbin/2;
    //    else
    //      j=i-c.nbin/2;

    f=-0.5*df+df*(float) i/(float) (c.nbin-1);

    rt=-f*f*s/((f0+f)*f0*f0);
    t=1.0/sqrt(1.0+pow((f/(0.47*df)),80));

    c.c[i].x=cos(rt)*t;
    c.c[i].y=sin(rt)*t;
  }

  return c;
}

// Compute chirp                                                                
struct chirp get_channel_chirp(double f0,double df,float dm,int nc)
{
  int i,k,l,m,nexp;
  float tdm;
  float s,rt,t,f,td1,td2,fc0,dfc;
  struct chirp c;

  // Compute dispersion sweep
  td1=dm*(pow(f0,-2)-pow(f0+0.5*df,-2))/DMCONSTANT;
  td2=dm*(pow(f0-0.5*df,-2)-pow(f0,-2))/DMCONSTANT;
  c.nd1=(int) floor(td1*df);
  c.nd2=(int) floor(td2*df);
  tdm=dm*(pow(f0-0.5*df,-2)-pow(f0+0.5*df,-2))/DMCONSTANT;
  c.nd=(int) floor(tdm*df);
  //  c.nd=c.nd1+c.nd2;                                                         
  nexp=(int) ceil(log(c.nd)/log(2.0))+1;
  //////////////////// HARDCODED DISPERSION KERNEL //////////////////////////           
  nexp=19;
  c.nd1=16384;                                                                  
  c.nd2=16384;                                                                  
  //c.nd1=65536;
  //  c.nd2=65536;
  c.nd=c.nd1+c.nd2;                                                             
  c.nbin=(int) pow(2.0,nexp);
  nexp=(int) ceil(log(c.nd2)/log(2.0));

  s=2.0*M_PI*dm/DMCONSTANT;

  // Number of channels per subband
  m=c.nbin/nc;

  // Allocate                                                                   
  c.c=(hipfftComplex *) malloc(sizeof(hipfftComplex)*c.nbin);

  dfc=df/nc;

  // Loop over subbands
  for (k=0;k<nc;k++) {
    fc0=f0-0.5*df+df*(float) k/(float) nc+0.5*df/(float) nc;
    for (i=0;i<m;i++) {
      f=-0.5*dfc+dfc*(float) i/(float) (m-1);

      rt=-f*f*s/((fc0+f)*fc0*fc0);
      t=1.0/sqrt(1.0+pow((f/(0.47*dfc)),80));

      l=i+k*m;
      
      c.c[l].x=cos(rt)*t;
      c.c[l].y=sin(rt)*t;
    }
  }
  /*
  // Compute chirp                                                              
  for (i=0;i<c.nbin;i++) {
    if (i<c.nbin/2)
      j=i+c.nbin/2;
    else
      j=i-c.nbin/2;

    f=-0.5*df+df*(float) j/(float) (c.nbin-1);

    rt=-f*f*s/((f0+f)*f0*f0);
    t=1.0/sqrt(1.0+pow((f/(0.47*df)),80));

    c.c[i].x=cos(rt)*t;
    c.c[i].y=sin(rt)*t;
  }
  */
  return c;
}

static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a,float s)
{
  hipfftComplex c;
  c.x=s*a.x;
  c.y=s*a.y;
  return c;
}

static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a,hipfftComplex b)
{
  hipfftComplex c;
  c.x=a.x*b.x-a.y*b.y;
  c.y=a.x*b.y+a.y*b.x;
  return c;
}

// Pointwise complex multiplication (and scaling)                               
static __global__ void PointwiseComplexMultiply(hipfftComplex *a,hipfftComplex *b,hipfftComplex *c,int nx,int ny,float scale)
{
  int i,j,k;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<nx && j<ny) {
    k=i+nx*j;
    c[k]=ComplexScale(ComplexMul(a[k],b[i]),scale);
  }
}

__global__ void unpack_and_padd(char *dbuf,int n,int nx,int ny,int i0,int m,hipfftComplex *cp1,hipfftComplex *cp2)
{
  int64_t i,j,k,l;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx && j<ny) {
    k=i+nx*j;
    l=i+m*j-i0;
    if (l<0 || l>=n) {
      cp1[k].x=0.0;
      cp1[k].y=0.0;
      cp2[k].x=0.0;
      cp2[k].y=0.0;
    } else {
      cp1[k].x=(float) dbuf[4*l];
      cp1[k].y=(float) dbuf[4*l+1];
      cp2[k].x=(float) dbuf[4*l+2];
      cp2[k].y=(float) dbuf[4*l+3];
    }
  }

  return;
}

__global__ void swap_spectrum_halves(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny)
{
  int64_t i,j,k,l,m;
  hipfftComplex tp1,tp2;

  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<nx/2 && j<ny) {
    if (i<nx/2)
      k=i+nx/2;
    else
      k=i-nx/2;
    l=i+nx*j;
    m=k+nx*j;
    tp1.x=cp1[l].x;
    tp1.y=cp1[l].y;
    tp2.x=cp2[l].x;
    tp2.y=cp2[l].y;
    cp1[l].x=cp1[m].x;
    cp1[l].y=cp1[m].y;
    cp2[l].x=cp2[m].x;
    cp2[l].y=cp2[m].y;
    cp1[m].x=tp1.x;
    cp1[m].y=tp1.y;
    cp2[m].x=tp2.x;
    cp2[m].y=tp2.y;
  }

  return;
}

__global__ void transpose_unpadd_and_detect(hipfftComplex *cp1,hipfftComplex *cp2,int nx,int ny,int nz,int i0,int i1,int n,float *fbuf)
{
  int64_t i,j,k,l,m,ii,jj;
  
  i=blockIdx.x*blockDim.x+threadIdx.x;
  j=blockIdx.y*blockDim.y+threadIdx.y;
  k=blockIdx.z*blockDim.z+threadIdx.z;
  if (i<nx && j<ny && k<nz) {
    m=i+nx*(j+ny*k);

    // Time index
    ii=i+(nx-i0-i1)*k-i0;

    // Frequency index
    jj=ny-j-1;

    // Array index
    l=jj+ny*ii;

    if (i>=i0 && i<=nx-i1 && ii>=0 && ii<n)
      fbuf[l]=sqrt(cp1[m].x*cp1[m].x+cp1[m].y*cp1[m].y+cp2[m].x*cp2[m].x+cp2[m].y*cp2[m].y);
  }

  return;
}

int main(int argc,char *argv[])
{
  int nsamp,nx,ny,nz,mx,my,mz,m,nchan=8;
  int iblock,nread;
  char *header,*hbuf,*dbuf;
  FILE *file,*ofile;
  float *fbuf,*dfbuf;
  hipfftComplex *cp1,*cp2,*dc;
  hipfftHandle ftc2cf,ftc2cb;
  int idist,odist,iembed,oembed,istride,ostride;
  dim3 blocksize,gridsize;
  struct chirp c;

  // Compute chirp
  c=get_channel_chirp(119.7265625,0.1953125,39.659298,nchan);

  nsamp=195312.5;
  nsamp*=600;
  nsamp=491520;

  // Data size
  nx=c.nbin;
  m=c.nbin-c.nd;
  ny=1;
  nz=(int) ceil(nsamp/(float) (m*ny));
  my=nchan;
  mx=nx/my;
  mz=nz;
  printf("%dx%dx%d %dx%dx%d %d\n",nx,ny,nz,mx,my,mz,m);

  // Allocate memory for complex timeseries
  checkCudaErrors(hipMalloc((void **) &cp1,sizeof(hipfftComplex)*nx*ny*nz));
  checkCudaErrors(hipMalloc((void **) &cp2,sizeof(hipfftComplex)*nx*ny*nz));

  // Allocate device memory for chirp                                                                
  checkCudaErrors(hipMalloc((void **) &dc,sizeof(hipfftComplex)*nx));

  // Allocate memory for redigitized output and header
  header=(char *) malloc(sizeof(char)*HEADERSIZE);
  hbuf=(char *) malloc(sizeof(char)*4*nsamp);
  checkCudaErrors(hipMalloc((void **) &dbuf,sizeof(char)*4*nsamp));

  // Allocate output buffers
  fbuf=(float *) malloc(sizeof(float)*nsamp);
  checkCudaErrors(hipMalloc((void **) &dfbuf,sizeof(float)*nsamp));

  // Generate FFT plan (batch in-place forward FFT)
  idist=nx;  odist=nx;  iembed=nx;  oembed=nx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cf,1,&nx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,ny*nz));

  // Generate FFT plan (batch in-place backward FFT)
  idist=mx;  odist=mx;  iembed=mx;  oembed=mx;  istride=1;  ostride=1;
  checkCudaErrors(hipfftPlanMany(&ftc2cb,1,&mx,&iembed,istride,idist,&oembed,ostride,odist,HIPFFT_C2C,my*mz));

  // Copy chirp to device                                                                            
  checkCudaErrors(hipMemcpy(dc,c.c,sizeof(hipfftComplex)*nx,hipMemcpyHostToDevice));

  // Read fil file header and dump in output file
  file=fopen("header.fil","r");
  fread(header,sizeof(char),351,file);
  fclose(file);
  ofile=fopen("test.fil","w");
  fwrite(header,sizeof(char),351,ofile);

  // Read file and buffer
  file=fopen("single_subband.dada","r");
  fread(header,sizeof(char),HEADERSIZE,file);

  // Loop over input file contents
  for (iblock=0;;iblock++) {
    nread=fread(hbuf,sizeof(char),4*nsamp,file)/4;
    printf("%d %d %d\n",iblock,nread,4*nsamp);
    if (nread==0)
      break;

    // Copy buffer to device
    checkCudaErrors(hipMemcpy(dbuf,hbuf,sizeof(char)*4*nread,hipMemcpyHostToDevice));

    // Unpack data and padd data
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=nx/blocksize.x+1;
    gridsize.y=nz/blocksize.y+1;
    gridsize.z=1;
    unpack_and_padd<<<gridsize,blocksize>>>(dbuf,nread,nx,nz,c.nd1,m,cp1,cp2);
    
    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_FORWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cf,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_FORWARD));
    
    // Swap spectrum halves for large FFTs
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx*my/blocksize.x+1;
    gridsize.y=mz/blocksize.y+1;
    gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mx*my,mz);
    
    // Perform complex multiplication of FFT'ed data with chirp (in place)                             
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=nx/blocksize.x+1;
    gridsize.y=nz/blocksize.y+1;
    gridsize.z=1;
    PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp1,dc,cp1,nx,nz,1.0/(float) nx);
    PointwiseComplexMultiply<<<gridsize,blocksize>>>(cp2,dc,cp2,nx,nz,1.0/(float) nx);
    
    // Swap spectrum halves for small FFTs
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx/blocksize.x+1;
    gridsize.y=my*mz/blocksize.y+1;
    gridsize.z=1;
    swap_spectrum_halves<<<gridsize,blocksize>>>(cp1,cp2,mx,my*mz);
    
    // Perform FFTs
    checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp1,(hipfftComplex *) cp1,HIPFFT_BACKWARD));
    checkCudaErrors(hipfftExecC2C(ftc2cb,(hipfftComplex *) cp2,(hipfftComplex *) cp2,HIPFFT_BACKWARD));
    
    // Detect data
    blocksize.x=32;
    blocksize.y=32;
    blocksize.z=1;
    gridsize.x=mx/blocksize.x+1;
    gridsize.y=my/blocksize.y+1;
    gridsize.z=mz/blocksize.z+1;
    transpose_unpadd_and_detect<<<gridsize,blocksize>>>(cp1,cp2,mx,my,mz,c.nd1/my,c.nd2/my,nread/my,dfbuf);
    
    // Copy buffer to host
    checkCudaErrors(hipMemcpy(fbuf,dfbuf,sizeof(float)*nread,hipMemcpyDeviceToHost));
    
    // Write buffer
    fwrite(fbuf,sizeof(float),nread,ofile);

    break;
  }

  // Close files
  fclose(ofile);
  fclose(file);

  // Free
  free(header);
  free(hbuf);
  free(fbuf);
  free(c.c);
  hipFree(dbuf);
  hipFree(dfbuf);
  hipFree(cp1);
  hipFree(cp2);
  hipFree(dc);

  // Free plan
  hipfftDestroy(ftc2cf);
  hipfftDestroy(ftc2cb);

  return 0;
}
